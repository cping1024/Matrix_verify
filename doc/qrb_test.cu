#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "glog/logging.h"
#include <sys/time.h>
static double getTimeOfMSeconds() {
        struct timeval tv;
        gettimeofday(&tv, NULL);
        return tv.tv_sec*1000. + tv.tv_usec/1000.;
}

void  init_array(float *a, const int N);
void  init_mat(float *a, const int N, const int M);
float diff_array(float* a, float* b, const int N);
float diff_mat(float* a, float* b, const int N, const int M);
void  showHelp(const int argc, const char** argv);
void  runTest(int argc, const char* argv);

int main(int argc, const char* argv){
    bool bTestResult = false;

    /// start the log
    fprintf(stderr, );
}


int main(int argc, char* argv[]) {

        hipblasStatus_t stat;
        hipblasHandle_t handle;
        stat = hipblasCreate(&handle);
        if (HIPBLAS_STATUS_SUCCESS != stat) {
                std::cout << "hipblasCreate Error." << std::endl;
                return 0;
        }

        const unsigned int M = 10000000;
        const unsigned int N = 160;
        float totalsz = (float)(M * (N + 1)) / 1204;
        totalsz *= 4; totalsz /= 1024; totalsz /=1024;
        fprintf(stderr, "Need tatal %f GB memory\n", totalsz);

        float* deviceMatrix;
        hipMalloc((void**)&deviceMatrix, sizeof(float)*M*N);
        float* deviceVector;
        hipMalloc((void**)&deviceVector, sizeof(float)*N);

        float* hostMatrix = new float[M*N];
        for (unsigned int i = 0; i < M*N; ++i) {
                hostMatrix[i] = 1.0 * rand() / RAND_MAX;
        }
        float* hostVector = new float[N];
        for (unsigned int i = 0; i < N; ++i) {
                hostVector[i] = 1.0* rand() / RAND_MAX;
        }
        hipMemcpy(deviceMatrix, hostMatrix, sizeof(float)*M*N, hipMemcpyHostToDevice);        

        float* deviceOut;
        hipMalloc((void**)&deviceOut, sizeof(float)*M);
        hipMemset(deviceOut, 0, sizeof(float)*M);

        hipblasOperation_t trans = HIPBLAS_OP_N;
        float alpha = 1.0;
        float beta = 0.0;
        double st = getTimeOfMSeconds();
        std::cout << "start sgemv\n";

        hipMemcpy(deviceVector, hostVector, sizeof(float)*N, hipMemcpyHostToDevice);

        stat = hipblasSgemv(handle, trans, M, N, &alpha, deviceMatrix, M, deviceVector, 1, &beta, deviceOut, 1);
        hipDeviceSynchronize();
        double et = getTimeOfMSeconds();
        std::cout << "hipblasSgemv time " << et-st << "ms" << std::endl;

        thrust::counting_iterator<int> iter(0);
        thrust::device_vector<int> indices(M);
        thrust::copy(iter, iter+indices.size(), indices.begin());
        thrust::device_ptr<float> keys( deviceOut );

        st = getTimeOfMSeconds();
        thrust::sort_by_key(keys, keys+M, indices.begin(), thrust::greater<float>());
        hipDeviceSynchronize();
        et = getTimeOfMSeconds();
        std::cout << "sort_by_key use time " << et-st << "ms" << std::endl;

        hipFree(deviceOut);
        hipFree(deviceVector);
        hipFree(deviceMatrix);

        delete[] hostVector;
        delete[] hostMatrix;
        return 0;
}

